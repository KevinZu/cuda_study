#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
using namespace std;



void MatrixMul_host(float *a, int a_rows, int a_cols, float *b, int b_rows, int b_cols, float *c) {
    for (int i = 0; i < a_rows; i++) {
        for (int j = 0; j < b_cols; j++) {
            float t = 0;
            for (int k = 0; k < b_rows; k++) {
                t += a[i*a_cols+k]*b[k*b_cols+j];
            }
            c[i*b_cols+j] = t;
        }
    }
}


void MatrixRandBin(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            if ((float)rand()/RAND_MAX > 0.5) {
                mat[i*cols+j] = 1.0f;
            }else {
                mat[i*cols+j] = -1.0f;
            }
        }
    }
}


float MatrixCompare(float *a,float *b,int rows,int cols){
    float err=0;
    for (int i=0;i<rows;i++){
        for (int j=0;j<cols;j++){
            err+=abs(a[i*cols+j]-b[i*cols+j]);  
        }
    }
    return err;
}


__global__ void MatrixMul_device(float *a, int a_rows, int a_cols, float *b, int b_rows, int b_cols, float *c) {
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int bix = blockIdx.x;
    int biy = blockIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int gdx = gridDim.x;
    int gdy = gridDim.y;

    for (int i = tix; i < b_cols; i += bdx) {
        float sum = 0;
        for (int k = 0; k < a_cols; k++) {
            sum += a[bix*a_rows+k]*b[k*b_cols+i];
        }
        c[bix*a_cols+i] = sum;
    }
}



int main()
{
	int Matrixsize=1000;
	float *a_host;
	float *a_device;
	float *b_host;
	float *b_device;
	float *result_host;
	float *result_device;
	float *result_cpu;
	a_host = (float*) malloc(sizeof(float) * Matrixsize * Matrixsize);
	b_host = (float*) malloc(sizeof(float) * Matrixsize * Matrixsize);
	result_host = (float*) malloc(sizeof(float) * Matrixsize * Matrixsize);
	result_cpu = (float*) malloc(sizeof(float) * Matrixsize * Matrixsize);
	srand(0);
	MatrixRandBin(a_host,Matrixsize,Matrixsize);
	MatrixRandBin(b_host,Matrixsize,Matrixsize);
	hipMalloc((void**)&a_device,sizeof(float) *Matrixsize * Matrixsize);
	hipMalloc((void**)&b_device,sizeof(float) *Matrixsize * Matrixsize);
	hipMalloc((void**)&result_device,sizeof(float) *Matrixsize * Matrixsize);
	hipMemcpy(a_device,a_host,sizeof(float) *Matrixsize * Matrixsize,hipMemcpyHostToDevice);
	hipMemcpy(b_device,b_host,sizeof(float) *Matrixsize * Matrixsize,hipMemcpyHostToDevice);


	hipEvent_t start_device, stop_device;
	float time_device;
	hipEventCreate(&start_device);
	hipEventCreate(&stop_device);
	hipEventRecord( start_device, 0 );
	dim3 gridsize(1000,1,1);
	dim3 blocksize(256,1,1);
	MatrixMul_device<<<gridsize,blocksize>>>(a_device,Matrixsize,Matrixsize,b_device,Matrixsize,Matrixsize,result_device);
	hipEventRecord( stop_device, 0 );
	hipEventSynchronize( stop_device );
	hipEventElapsedTime( &time_device, start_device, stop_device );
	hipEventDestroy( start_device );
	hipEventDestroy( stop_device );
	cout<<"gputime="<<time_device<<"ms"<<endl;

	hipMemcpy(result_host, result_device,sizeof(float) *Matrixsize * Matrixsize,hipMemcpyDeviceToHost);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(result_device);
	clock_t start_host = clock();
	MatrixMul_host(a_host,Matrixsize,Matrixsize,b_host,Matrixsize,Matrixsize,result_cpu);
	cout<<"cputime="<<(double)(clock() - start_host)/1000<<"ms"<<endl;
	float err=MatrixCompare(result_cpu,result_host,Matrixsize,Matrixsize);
	cout<<"err in gpu and cpu = "<<err<<endl;
}

